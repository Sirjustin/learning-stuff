//
// Copyright (c) Microsoft. All rights reserved.
// Licensed under the MIT license. See LICENSE.md file in the project root for full license information.
//

#include "stdafx.h"
#include "BestGpu.h"

#ifndef CPUONLY

#include "GPUWatcher.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

int GPUWatcher::GetGPUIdWithTheMostFreeMemory()
{
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess || deviceCount == 0)
    {
        return -1;
    }
    int curDev = 0;
    size_t curMemory = 0;
    for (int dev = 0; dev < deviceCount; ++dev)
    {
        size_t freeMem = GetFreeMemoryOnCUDADevice(dev);
        if (freeMem > curMemory)
        {
            curMemory = freeMem;
            curDev = dev;
        }
    }
    return curDev;
}

size_t GPUWatcher::GetFreeMemoryOnCUDADevice(int devId)
{
    hipError_t result = hipSetDevice(devId);
    if (result != hipSuccess)
    {
        return 0;
    }
    // get the amount of free memory on the graphics card
    size_t free = 0;
    size_t total = 0;
    result = hipMemGetInfo(&free, &total);
    if (result != hipSuccess)
    {
        return 0;
    }
    else
        return free;
}

GPUWatcher::GPUWatcher(void)
{
}

GPUWatcher::~GPUWatcher(void)
{
}

#endif // CPUONLY
